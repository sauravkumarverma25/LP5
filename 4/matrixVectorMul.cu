
#include <hip/hip_runtime.h>
#include <time.h>

#include <cmath>
#include <cstdlib>
#include <iostream>

#define checkCudaErrors(call)                                                                 \
    do {                                                                                      \
        hipError_t err = call;                                                               \
        if (err != hipSuccess) {                                                             \
            printf("CUDA error at %s %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE);                                                               \
        }                                                                                     \
    } while (0)

using namespace std;

__global__ void matrixVectorMultiplication(int *a, int *b, int *c, int n) {
    int row = threadIdx.x + blockDim.x * blockIdx.x;
    int sum = 0;

    if (row < n)
        for (int j = 0; j < n; j++) {
            sum = sum + a[row * n + j] * b[j];
        }

    c[row] = sum;
}

int main() {
    int *a, *b, *c;
    int *a_dev, *b_dev, *c_dev;
    int n = 10;

    a = new int[n * n];
    b = new int[n];
    c = new int[n];
    int *d = new int[n];
    int size = n * sizeof(int);
    checkCudaErrors(hipMalloc(&a_dev, size * size));
    checkCudaErrors(hipMalloc(&b_dev, size));
    checkCudaErrors(hipMalloc(&c_dev, size));

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            a[i * n + j] = rand() % 10;
        }
        b[i] = rand() % 10;
    }

    cout << "Given matrix is =>\n";
    for (int row = 0; row < n; row++) {
        for (int col = 0; col < n; col++) {
            cout << a[row * n + col] << " ";
        }
        cout << "\n";
    }
    cout << "\n";

    cout << "Given vector is =>\n";
    for (int i = 0; i < n; i++) {
        cout << b[i] << ", ";
    }
    cout << "\n\n";

    hipEvent_t start, end;

    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&end));

    checkCudaErrors(hipMemcpy(a_dev, a, size * size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(b_dev, b, size, hipMemcpyHostToDevice));

    dim3 threadsPerBlock(n, n);
    dim3 blocksPerGrid(1, 1);

    checkCudaErrors(hipEventRecord(start));
    matrixVectorMultiplication<<<blocksPerGrid, threadsPerBlock>>>(a_dev, b_dev, c_dev, n);

    checkCudaErrors(hipEventRecord(end));
    checkCudaErrors(hipEventSynchronize(end));

    float time = 0.0;
    checkCudaErrors(hipEventElapsedTime(&time, start, end));

    checkCudaErrors(hipMemcpy(c, c_dev, size, hipMemcpyDeviceToHost));

    // CPU matrixVector multiplication
    int sum = 0;
    for (int row = 0; row < n; row++) {
        sum = 0;
        for (int col = 0; col < n; col++) {
            sum = sum + a[row * n + col] * b[col];
        }
        d[row] = sum;
    }

    cout << "CPU product is =>\n";
    for (int i = 0; i < n; i++) {
        cout << d[i] << ", ";
    }
    cout << "\n\n";

    cout << "GPU product is =>\n";
    for (int i = 0; i < n; i++) {
        cout << c[i] << ", ";
    }
    cout << "\n\n";

    int error = 0;
    for (int i = 0; i < n; i++) {
        error += d[i] - c[i];
        if (0 != (d[i] - c[i])) {
            cout << "Error at (" << i << ") => GPU: " << c[i] << ", CPU: " << d[i] << "\n";
        }
    }

    cout << "Error: " << error;
    cout << "\nTime Elapsed: " << time;

    return 0;
}

/*

OUTPUT:

Given matrix is =>
3 6 7 5 3 5 6 2 9 1
7 0 9 3 6 0 6 2 6 1
7 9 2 0 2 3 7 5 9 2
8 9 7 3 6 1 2 9 3 1
4 7 8 4 5 0 3 6 1 0
3 2 0 6 1 5 5 4 7 6
6 9 3 7 4 5 2 5 4 7
4 3 0 7 8 6 8 8 4 3
4 9 2 0 6 8 9 2 6 6
9 5 0 4 8 7 1 7 2 7

Given vector is =>
2, 8, 2, 9, 6, 5, 4, 1, 4, 2,

CPU product is =>
220, 147, 190, 201, 168, 171, 245, 235, 234, 210,

GPU product is =>
220, 147, 190, 201, 168, 171, 245, 235, 234, 210,

Error: 0
Time Elapsed: 0.014336

*/
