
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__ void vecAdd(double *a, double *b, double *c, int n)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
        c[id] = a[id] + b[id];
}

int main()
{
    int n = 10;
    size_t bytes = n * sizeof(double);
    double *h_a = (double *)malloc(bytes), *h_b = (double *)malloc(bytes), *h_c = (double *)malloc(bytes);
    double *d_a, *d_b, *d_c;

    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    for (int i = 0; i < n; i++)
    {
        h_a[i] = sin(i) * sin(i);
        h_b[i] = cos(i) * cos(i);
    }

    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

    vecAdd<<<(int)ceil((float)n / 1024), 1024>>>(d_a, d_b, d_c, n);

    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

    double sum = 0;
    for (int i = 0; i < n; i++)
    {
        sum += h_c[i];
        printf("%f + %f = %f\n", h_a[i], h_b[i], h_c[i]);
    }

    printf("final result: %.2f\n", sum);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}